#include <hip/hip_fp16.h>
#include <hipblas.h>
#include "linear.h"

bool has_handle = false;
hipblasHandle_t handle;

void cublas_init_handle(){
	hipblasStatus_t status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		std::cerr << "cublas_init_handle failed: " << status << std::endl;
		throw std::runtime_error("cublas_init_handle failed");
	}
}

/* Row major 
 *   A (m x k) einsum(ik, jk -> ij) B (n x k) = C (m x n) 
 * Equivalent to column major 
 *   B (n x k) @ A^T (k x m) = C^T (m x n)
 */
void array_linear(
	int m,
	int n,
	int k,
	const half* Aarray,
	const half* Barray,
	half* Carray
) {
  const float alpha = 1.0;
  const float beta = 0.0;
	hipblasStatus_t status = cublasSgemmEx(
		handle,
		HIPBLAS_OP_T,
		HIPBLAS_OP_N,
		n,
		m,
		k,
		&alpha,
		Barray,
		HIP_R_16F,
		k,
		Aarray,
		HIP_R_16F,
    k,
		&beta,
		Carray,
		HIP_R_16F,
		n
	);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		std::cerr << "hipblasGemmEx failed: " << status << std::endl;
		throw std::runtime_error("hipblasGemmEx failed");
	}
}

torch::Tensor linear(
  torch::Tensor a,
  torch::Tensor w
) {
  int m = a.size(0);
  int k = a.size(1);
  int n = w.size(0);
	auto r = torch::empty({m, n}, a.options());

  const half* a_data = (half*)a.data_ptr();
	const half* w_data = (half*)w.data_ptr();
	half* r_data = (half*)r.data_ptr();

	if (!has_handle) {
		cublas_init_handle();
		has_handle = true;
	}

	array_linear(
		m,
		n,
		k,
		a_data,
		w_data,
		r_data
	);

	return r;
}